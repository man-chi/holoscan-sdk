#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <unistd.h>
#include <cub/device/device_reduce.cuh>

#include <functional>
#include <memory>
#include <string>
#include <utility>
#include <vector>

#include <iostream>

#include "data_processor.hpp"
#include "holoinfer.hpp"
#include "holoinfer_constants.hpp"

namespace holoscan {
namespace inference {

// Green context was introduced with CUDA 12.4, it is not supported by all driver versions.
// Therefore dynamically get the symbol to avoid runtime link errors when the symbol is not
// exposed by libcuda.so.
static hipError_t (*fnCuStreamGetGreenCtx)(hipStream_t, CUgreenCtx*) = nullptr;
static hipError_t (*fnCuCtxFromGreenCtx)(hipCtx_t*, CUgreenCtx) = nullptr;

/**
 * This class implements an iterator which skips `step` elements between each iteration.
 */
class step_iterator {
 public:
  using iterator_category = std::random_access_iterator_tag;
  using value_type = float;
  using difference_type = ptrdiff_t;
  using pointer = float*;
  using reference = float&;

  explicit __host__ __device__ __forceinline__ step_iterator(pointer cur, size_t step)
      : cur_(cur), step_(step) {}

  template <typename Distance>
  __host__ __device__ __forceinline__ reference operator[](Distance offset) const {
    return cur_[offset * step_];
  }

  template <typename Distance>
  __host__ __device__ __forceinline__ step_iterator operator+(Distance n) {
    step_iterator retval(cur_ + n * step_, step_);
    return retval;
  }

 private:
  pointer cur_;
  size_t step_;
};

#if CUB_VERSION >= 280000
// Type aliases and index extraction for CUB 2.8.0+
using cubResultType = cuda::std::int64_t;
#define GET_INDEX(result, idx) ((result)[(idx)])
#else
// Type aliases and index extraction for older CUB versions
using cubResultType = hipcub::KeyValuePair<int, float>;
#define GET_INDEX(result, idx) ((result)[(idx)].key)
#endif  // CUB_VERSION >= 280000

/**
 * CUDA kernel normalizing the coordinates stored in the result array.
 * Works with both CUB versions through conditional index extraction.
 *
 * @param rows
 * @param cols
 * @param channels
 * @param d_result - either d_index (CUB 2.8.0+) or d_argmax (older CUB)
 * @param out
 */
static __global__ void normalize(size_t rows, size_t cols, size_t channels, cubResultType* d_result,
                                 float* out) {
  const uint index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= channels) {
    return;
  }

  const int src_index = GET_INDEX(d_result, index);
  int row = src_index / cols;
  int col = src_index - (row * cols);
  out[index * 2 + 0] = (float)row / (float)rows;
  out[index * 2 + 1] = (float)col / (float)cols;
}

void DataProcessor::max_per_channel_scaled_cuda(size_t rows, size_t cols, size_t channels,
                                                const float* indata, float* outdata,
                                                hipStream_t hip_stream) {
  /// @todo This algorithm needs temporary storage, currently data processors are just functions
  /// without state. This should be an object with state so we can avoid re-allocating the temporary
  /// storage at each invocation.

#if CUB_VERSION >= 280000
  // Allocate result storage
  float* d_max_out = nullptr;
  cuda::std::int64_t* d_index_out = nullptr;
  check_cuda(hipMallocAsync(&d_max_out, sizeof(float) * channels, hip_stream));
  check_cuda(hipMallocAsync(&d_index_out, sizeof(cuda::std::int64_t) * channels, hip_stream));
#else
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wdeprecated-declarations"
  hipcub::KeyValuePair<int, float>* d_argmax = nullptr;
  check_cuda(
      hipMallocAsync(&d_argmax, sizeof(hipcub::KeyValuePair<int, float>) * channels, hip_stream));
#endif

  // get temp storage size
  void* d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::ArgMax(d_temp_storage,
                            temp_storage_bytes,
                            indata,
#if CUB_VERSION >= 280000
                            d_max_out,
                            d_index_out,
#else
                            d_argmax,
#endif
                            rows * cols);

  // Allocate temporary storage
  check_cuda(hipMallocAsync(&d_temp_storage, temp_storage_bytes, hip_stream));

  for (size_t channel = 0; channel < channels; ++channel) {
    step_iterator iterator((float*)(indata + channel), channels);
    hipcub::DeviceReduce::ArgMax(d_temp_storage,
                              temp_storage_bytes,
                              iterator,
#if CUB_VERSION >= 280000
                              &d_max_out[channel],
                              &d_index_out[channel],
#else
                              &d_argmax[channel],
#endif
                              rows * cols,
                              hip_stream);
  }

  check_cuda(hipFreeAsync(d_temp_storage, hip_stream));

  dim3 block(32, 1, 1);
  dim3 grid((channels + block.x - 1) / block.x, 1, 1);
  normalize<<<grid, block, 0, hip_stream>>>(rows,
                                             cols,
                                             channels,
#if CUB_VERSION >= 280000
                                             d_index_out,
#else
                                             d_argmax,
#endif
                                             outdata);
  check_cuda(hipPeekAtLastError());

#if CUB_VERSION >= 280000
  check_cuda(hipFreeAsync(d_max_out, hip_stream));
  check_cuda(hipFreeAsync(d_index_out, hip_stream));
#else
  check_cuda(hipFreeAsync(d_argmax, hip_stream));
#pragma GCC diagnostic pop
#endif
}

InferStatus DataProcessor::launchCustomKernel(const std::vector<std::string>& ids,
                                              const std::vector<int>& dimensions, const void* input,
                                              std::vector<int64_t>& processed_dims,
                                              DataMap& processed_data_map,
                                              const std::vector<std::string>& output_tensors,
                                              bool process_with_cuda, hipStream_t hip_stream) {
  if (output_tensors.size() == 0) {
    return InferStatus(holoinfer_code::H_ERROR,
                       "Data processor, Output tensor size 0 in launchCustomKernel.");
  }

  if (!process_with_cuda) {
    return InferStatus(holoinfer_code::H_ERROR,
                       "Data processor, launching custom kernel must have I/O on cuda.");
  }

  auto out_tensor_name = output_tensors[0];

  if (cuda_graph_created_.find(out_tensor_name) == cuda_graph_created_.end()) {
    cuda_graph_created_[out_tensor_name] = false;
    cuda_graph_instantiated_[out_tensor_name] = false;
  }
  size_t dsize = accumulate(dimensions.begin(), dimensions.end(), 1, std::multiplies<size_t>());
  size_t dimensionX = dimensions[0];
  size_t dimensionY = dimensions[1];
  size_t dimensionZ = dimensions[2];

  int kernel_count = ids.size();
  const char* error_string;

  // Check if the cuda stream is associated with a green context
  hipCtx_t context = nullptr;
  hipError_t result;

  if (fnCuStreamGetGreenCtx) {
    CUgreenCtx green_ctx = nullptr;

    result = fnCuStreamGetGreenCtx(hip_stream, &green_ctx);
    if (result != hipSuccess) {
      hipDrvGetErrorString(result, &error_string);
      HOLOSCAN_LOG_ERROR("CUDA stream get green context failed in launchKernel: {}", error_string);
      return InferStatus(holoinfer_code::H_ERROR,
                         "Data processor, CUDA stream get green context failed.");
    }

    if (green_ctx && fnCuCtxFromGreenCtx) {
      result = fnCuCtxFromGreenCtx(&context, green_ctx);
      if (result != hipSuccess) {
        hipDrvGetErrorString(result, &error_string);
        HOLOSCAN_LOG_ERROR("CUDA context from green context failed in launchKernel: {}",
                           error_string);
        return InferStatus(holoinfer_code::H_ERROR,
                           "Data processor, CUDA context from green context failed.");
      }
    }
  }
  if (!context) {
    result = cuStreamGetCtx(hip_stream, &context);
    if (result != hipSuccess) {
      hipDrvGetErrorString(result, &error_string);
      HOLOSCAN_LOG_ERROR("CUDA context from stream failed in launchKernel: {}", error_string);
      return InferStatus(holoinfer_code::H_ERROR,
                         "Data processor, Cuda context from stream failed.");
    }
  }

  // Successfully get the context from the stream
  result = hipCtxPushCurrent(context);
  if (result != hipSuccess) {
    hipDrvGetErrorString(result, &error_string);
    HOLOSCAN_LOG_ERROR("CUDA context push failed in launchKernel: {}", error_string);
    return InferStatus(holoinfer_code::H_ERROR,
                       "Data processor, CUDA context push failed with stream.");
  }

  if (first_time_kernel_launch_map_.find(out_tensor_name) == first_time_kernel_launch_map_.end()) {
    first_time_kernel_launch_map_[out_tensor_name] = true;
    intermediate_inputs_[out_tensor_name].push_back(const_cast<void*>(input));

    for (int i = 1; i < kernel_count; i++) {
      auto intermediate_buffer = std::make_shared<DataBuffer>(output_dtype_.at(ids[i - 1]));
      if (dynamic_output_dim_) {
        auto dyn_dimensions = custom_kernel_output_dimensions_.at(ids[i - 1]);
        dsize =
            accumulate(dyn_dimensions.begin(), dyn_dimensions.end(), 1, std::multiplies<size_t>());
      }
      intermediate_buffer->device_buffer_->resize(dsize);

      intermediate_buffers_[out_tensor_name].push_back(std::move(intermediate_buffer));
      intermediate_inputs_[out_tensor_name].push_back(
          intermediate_buffers_[out_tensor_name].back()->device_buffer_->data());
    }

    // create the output data

    if (processed_data_map.find(out_tensor_name) == processed_data_map.end()) {
      HOLOSCAN_LOG_INFO("Allocating memory for {} in launchGenericKernel", out_tensor_name);
      const auto [db, success] = processed_data_map.insert(
          {out_tensor_name, std::make_shared<DataBuffer>(output_dtype_.at(ids[kernel_count - 1]))});

      if (dynamic_output_dim_) {
        auto dyn_dimensions = custom_kernel_output_dimensions_.at(ids[kernel_count - 1]);
        dsize =
            accumulate(dyn_dimensions.begin(), dyn_dimensions.end(), 1, std::multiplies<size_t>());
        processed_dims.insert(processed_dims.begin(), dyn_dimensions.begin(), dyn_dimensions.end());
      } else {
        processed_dims.insert(processed_dims.begin(), dimensions.begin(), dimensions.end());
      }
      db->second->device_buffer_->resize(dsize);
      db->second->host_buffer_->resize(dsize);
    }

    intermediate_inputs_[out_tensor_name].push_back(
        processed_data_map.at(out_tensor_name)->device_buffer_->data());
  } else {
    intermediate_inputs_[out_tensor_name][0] = const_cast<void*>(input);
  }

  int buffer_count = 0;

  for (auto id : ids) {
    if (dynamic_output_dim_ && buffer_count > 0) {
      // output of the previous kernel is the input to the current kernel
      auto dyn_output_dimensions = custom_kernel_output_dimensions_.at(ids[buffer_count - 1]);
      if (dyn_output_dimensions.size() == 1) {
        dimensionX = dyn_output_dimensions[0];
        dimensionY = 1;
        dimensionZ = 1;
      } else if (dyn_output_dimensions.size() == 2) {
        dimensionX = dyn_output_dimensions[0];
        dimensionY = dyn_output_dimensions[1];
        dimensionZ = 1;
      } else if (dyn_output_dimensions.size() == 3) {
        dimensionX = dyn_output_dimensions[0];
        dimensionY = dyn_output_dimensions[1];
        dimensionZ = dyn_output_dimensions[2];
      }
      dsize = accumulate(
          dyn_output_dimensions.begin(), dyn_output_dimensions.end(), 1, std::multiplies<size_t>());
    } else {
      dsize = accumulate(dimensions.begin(), dimensions.end(), 1, std::multiplies<size_t>());
    }

    std::vector<void*> args = {&intermediate_inputs_[out_tensor_name][buffer_count],
                               &intermediate_inputs_[out_tensor_name][buffer_count + 1],
                               &dsize};

    // Find the dimensionality
    std::vector<std::string> threads_per_block;
    string_split(custom_kernel_thread_per_block_.at(id), threads_per_block, ',');

    int threadsPerBlockx = 1, threadsPerBlocky = 1, threadsPerBlockz = 1;
    int blocksPerGridx = 1, blocksPerGridy = 1, blocksPerGridz = 1;

    // compute appropriate grid and block size
    switch (threads_per_block.size()) {
      case 1:
      default: {
        threadsPerBlockx = std::atoi(threads_per_block[0].c_str());
        blocksPerGridx = (dsize + threadsPerBlockx - 1) / threadsPerBlockx;
        break;
      }

      case 2: {
        threadsPerBlockx = std::atoi(threads_per_block[0].c_str());
        threadsPerBlocky = std::atoi(threads_per_block[1].c_str());
        blocksPerGridx = (dimensions[0] + threadsPerBlockx - 1) / threadsPerBlockx;
        blocksPerGridy = (dimensions[1] + threadsPerBlocky - 1) / threadsPerBlocky;

        std::vector<void*> newargs = {&intermediate_inputs_[out_tensor_name][buffer_count],
                                      &intermediate_inputs_[out_tensor_name][buffer_count + 1],
                                      &dimensionX,
                                      &dimensionY};
        args = std::move(newargs);
        break;
      }

      case 3: {
        threadsPerBlockx = std::atoi(threads_per_block[0].c_str());
        threadsPerBlocky = std::atoi(threads_per_block[1].c_str());
        threadsPerBlockz = std::atoi(threads_per_block[2].c_str());
        blocksPerGridx = (dimensions[0] + threadsPerBlockx - 1) / threadsPerBlockx;
        blocksPerGridy = (dimensions[1] + threadsPerBlocky - 1) / threadsPerBlocky;
        blocksPerGridz = (dimensions[2] + threadsPerBlockz - 1) / threadsPerBlockz;
        std::vector<void*> newargs = {&intermediate_inputs_[out_tensor_name][buffer_count],
                                      &intermediate_inputs_[out_tensor_name][buffer_count + 1],
                                      &dimensionX,
                                      &dimensionY,
                                      &dimensionZ};
        args = std::move(newargs);
        break;
      }
    }

    dim3 gridDim(blocksPerGridx, blocksPerGridy, blocksPerGridz);
    dim3 blockDim(threadsPerBlockx, threadsPerBlocky, threadsPerBlockz);

    if (use_cuda_graph_) {
      if (!cuda_graph_created_[out_tensor_name]) {
        cuda_graph_created_[out_tensor_name] = true;
        hipGraph_t l_graph;
        result = hipGraphCreate(&l_graph, 0);
        if (result != hipSuccess) {
          hipDrvGetErrorString(result, &error_string);

          HOLOSCAN_LOG_ERROR("CUDA graph creation failed in launchKernel: {}", error_string);
          return InferStatus(holoinfer_code::H_ERROR,
                             "Data processor, CUDA graph creation failed.");
        }
        graph_[out_tensor_name] = l_graph;
      }

      if (!cuda_graph_instantiated_[out_tensor_name]) {
        hipKernelNodeParams kernelNodeParam = {0};
        kernelNodeParam.func = kernel_.at(id);
        kernelNodeParam.gridDimX = blocksPerGridx;
        kernelNodeParam.gridDimY = blocksPerGridy;
        kernelNodeParam.gridDimZ = blocksPerGridz;
        kernelNodeParam.blockDimX = threadsPerBlockx;
        kernelNodeParam.blockDimY = threadsPerBlocky;
        kernelNodeParam.blockDimZ = threadsPerBlockz;
        kernelNodeParam.sharedMemBytes = 0;
        kernelNodeParam.kernelParams = args.data();
        kernel_node_params_[out_tensor_name].push_back(std::move(kernelNodeParam));
        hipGraphNode_t kernelNode;
        if (buffer_count == 0) {
          result = hipGraphAddKernelNode(&kernelNode,
                                        graph_[out_tensor_name],
                                        nullptr,
                                        0,
                                        &kernel_node_params_[out_tensor_name][buffer_count]);
        } else {
          hipGraphNode_t dependencies[] = {kernel_nodes_[out_tensor_name][buffer_count - 1]};
          result = hipGraphAddKernelNode(&kernelNode,
                                        graph_[out_tensor_name],
                                        dependencies,
                                        1,
                                        &kernel_node_params_[out_tensor_name][buffer_count]);
        }
        if (result != hipSuccess) {
          hipDrvGetErrorString(result, &error_string);

          HOLOSCAN_LOG_ERROR("CUDA graph node addition failed in launchKernel: {}", error_string);
          return InferStatus(holoinfer_code::H_ERROR,
                             "Data processor, CUDA graph node creation failed.");
        }
        kernel_nodes_[out_tensor_name].push_back(kernelNode);
      }

      if (cuda_graph_instantiated_[out_tensor_name]) {
        hipKernelNodeParams updatedParams = kernel_node_params_[out_tensor_name][buffer_count];
        updatedParams.kernelParams = args.data();
        result = hipGraphExecKernelNodeSetParams(cuda_graph_instance_[out_tensor_name],
                                                kernel_nodes_[out_tensor_name][buffer_count],
                                                &updatedParams);
        if (result != hipSuccess) {
          hipDrvGetErrorString(result, &error_string);

          HOLOSCAN_LOG_ERROR("CUDA graph node setting failed in launchKernel: {}", error_string);
          return InferStatus(holoinfer_code::H_ERROR,
                             "Data processor, CUDA graph node setting failed.");
        }
      }
    } else {
      result = hipModuleLaunchKernel(kernel_.at(id),
                              blocksPerGridx,
                              blocksPerGridy,
                              blocksPerGridz,
                              threadsPerBlockx,
                              threadsPerBlocky,
                              threadsPerBlockz,
                              0,
                              reinterpret_cast<hipStream_t>(hip_stream),
                              args.data(),
                              0);

      if (result != hipSuccess) {
        hipDrvGetErrorString(result, &error_string);
        HOLOSCAN_LOG_ERROR("CUDA error in launching custom kernel: {}", error_string);
        return InferStatus(holoinfer_code::H_ERROR,
                           "Data processor, error in launching custom kernel.");
      }
    }

    buffer_count++;
  }

  if (use_cuda_graph_) {
    if (!cuda_graph_instantiated_[out_tensor_name]) {
      result =
          hipGraphInstantiate(&cuda_graph_instance_[out_tensor_name], graph_[out_tensor_name], 0);
      if (result != hipSuccess) {
        hipDrvGetErrorString(result, &error_string);

        HOLOSCAN_LOG_ERROR("CUDA graph instantiation failed in launchKernel: {}", error_string);
        return InferStatus(holoinfer_code::H_ERROR,
                           "Data processor, CUDA graph instantiation failed.");
      }
    }
    result = hipGraphLaunch(cuda_graph_instance_[out_tensor_name],
                           reinterpret_cast<hipStream_t>(hip_stream));
    if (result != hipSuccess) {
      hipDrvGetErrorString(result, &error_string);

      HOLOSCAN_LOG_ERROR("CUDA graph launch failed in launchKernel: {}", error_string);
      return InferStatus(holoinfer_code::H_ERROR, "Data processor, CUDA graph launch failed.");
    }

    cuda_graph_instantiated_[out_tensor_name] = true;
  }

  result = hipCtxPopCurrent(nullptr);
  if (result != hipSuccess) {
    hipDrvGetErrorString(result, &error_string);

    HOLOSCAN_LOG_ERROR("Cuda context pop failed in launchKernel: {}", error_string);
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, Cuda context setting failed.");
  }

  return InferStatus();
}

InferStatus DataProcessor::prepareCustomKernel() {
  hiprtcProgram prog;
  hiprtcResult nvResult =
      hiprtcCreateProgram(&prog, custom_cuda_src_.c_str(), "customKernel.cu", 0, NULL, NULL);

  if (nvResult != HIPRTC_SUCCESS) {
    HOLOSCAN_LOG_INFO("Error in NVRTC program creation {}", hiprtcGetErrorString(nvResult));
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, NVRTC program creation failed.");
  }

  if (prog == NULL) {
    HOLOSCAN_LOG_ERROR("Created program is NULL from nvrtc");
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, NVRTC created NULL program.");
  }

  int device;
  hipError_t err = hipGetDevice(&device);
  if (err != hipSuccess) {
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, Failed to get CUDA device id.");
  }

  hipDeviceProp_t prop;
  err = hipGetDeviceProperties(&prop, device);
  if (err != hipSuccess) {
    return InferStatus(holoinfer_code::H_ERROR,
                       "Data processor, Failed to get CUDA device properties.");
  }

  HOLOSCAN_LOG_DEBUG("Device {}: {}", device, prop.name);
  HOLOSCAN_LOG_DEBUG("GPU Compute Capability: {}.{}", prop.major, prop.minor);

  std::string arch_compile_string = "--gpu-architecture=sm_";
  arch_compile_string.append(std::to_string(prop.major)).append(std::to_string(prop.minor));
  HOLOSCAN_LOG_DEBUG("GPU architecture compilation flag: {}", arch_compile_string);

  const char* opts[] = {arch_compile_string.c_str()};

  hiprtcResult compileResult = hiprtcCompileProgram(prog, 1, opts);
  if (compileResult != HIPRTC_SUCCESS) {
    size_t logSize;
    hiprtcGetProgramLogSize(prog, &logSize);
    char* log = new char[logSize];
    hiprtcGetProgramLog(prog, log);
    HOLOSCAN_LOG_ERROR("Cuda kernel compilation error: {}", log);
    delete[] log;
    HOLOSCAN_LOG_ERROR("NVRTC compilation failed. Please review the custom cuda kernel.");
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, NVRTC compilation failed.");
  }

  size_t ptxSize;
  nvResult = hiprtcGetCodeSize(prog, &ptxSize);
  if (nvResult != HIPRTC_SUCCESS) {
    HOLOSCAN_LOG_ERROR("Error in NVRTC get ptx size {}", hiprtcGetErrorString(nvResult));
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, NVRTC get ptx size failed.");
  }

  HOLOSCAN_LOG_DEBUG("PTX size: {}", ptxSize);
  std::vector<char> ptx(ptxSize);
  nvResult = hiprtcGetCode(prog, ptx.data());
  if (nvResult != HIPRTC_SUCCESS) {
    HOLOSCAN_LOG_ERROR("Error in NVRTC ptx data {}", hiprtcGetErrorString(nvResult));
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, NVRTC get ptx failed.");
  }
  HOLOSCAN_LOG_DEBUG("PTX file: {}", ptx.data());
  HOLOSCAN_LOG_INFO("NVRTC kernel compilation succeeded.");

  nvResult = hiprtcDestroyProgram(&prog);
  if (nvResult != HIPRTC_SUCCESS) {
    HOLOSCAN_LOG_ERROR("Error in NVRTC program destruction. {}", hiprtcGetErrorString(nvResult));
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, NVRTC prog destruction failed.");
  }

  hipError_t result;
  result = hipInit(0);
  if (result != hipSuccess) {
    HOLOSCAN_LOG_ERROR("Cuda Init failed.");
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, Cuda Init failed.");
  }

  result = hipDeviceGet(&device_, 0);
  hipCtx_t context = nullptr;
  if (result == hipSuccess) {
    // Retain the device primary context so it matches streams from CudaStreamPool
    result = hipDevicePrimaryCtxRetain(&context, device_);
  }
  if (result != hipSuccess) {
    HOLOSCAN_LOG_ERROR("Cuda primary context retain failed.");
    return InferStatus(holoinfer_code::H_ERROR,
                       "Data processor, Cuda primary context retain failed.");
  }

  // Make the retained primary context current while loading module/functions
  result = hipCtxPushCurrent(context);
  if (result != hipSuccess) {
    HOLOSCAN_LOG_ERROR("Cuda Context push failed in prepareKernel.");
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, Cuda context push failed.");
  }

  result = hipModuleLoadData(&module_, ptx.data());
  if (result != hipSuccess) {
    const char* error_string;
    hipDrvGetErrorString(result, &error_string);
    HOLOSCAN_LOG_ERROR("Cuda module loading failed. {}", error_string);
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, Cuda module loading failed.");
  }

  for (const auto& item : custom_kernel_thread_per_block_) {
    auto kernel_id = item.first;
    auto kernel_name = "customKernel" + kernel_id;

    result = hipModuleGetFunction(&kernel_[kernel_id], module_, kernel_name.c_str());
    if (result != hipSuccess) {
      const char* error_string;
      hipDrvGetErrorString(result, &error_string);
      HOLOSCAN_LOG_ERROR("Cuda module get function failed. {}", error_string);
      return InferStatus(holoinfer_code::H_ERROR,
                         "Data processor, Cuda module get function failed.");
    }
  }

  // get CUDA green context functions
  static std::once_flag flag1;
  std::call_once(flag1, []() {
    const char* error_string;
    hipError_t result;

    int driver_version = 0;
    result = hipDriverGetVersion(&driver_version);
    if (result != hipSuccess) {
      hipDrvGetErrorString(result, &error_string);
      HOLOSCAN_LOG_ERROR("CUDA driver get version failed in prepareCustomKernel: {}", error_string);
    }
    result = hipGetProcAddress("cuStreamGetGreenCtx",
                              reinterpret_cast<void**>(&fnCuStreamGetGreenCtx),
                              driver_version,
                              0,
                              nullptr);
    if (result != hipSuccess) {
      hipDrvGetErrorString(result, &error_string);
      HOLOSCAN_LOG_ERROR(
          "CUDA get proc address of 'cuStreamGetGreenCtx' failed in prepareCustomKernel: {}",
          error_string);
    }
    result = hipGetProcAddress("cuCtxFromGreenCtx",
                              reinterpret_cast<void**>(&fnCuCtxFromGreenCtx),
                              driver_version,
                              0,
                              nullptr);
    if (result != hipSuccess) {
      hipDrvGetErrorString(result, &error_string);
      HOLOSCAN_LOG_ERROR(
          "CUDA get proc address of 'cuCtxFromGreenCtx' failed in prepareCustomKernel: {}",
          error_string);
    }
  });

  if (hipCtxPopCurrent(nullptr) != hipSuccess) {
    HOLOSCAN_LOG_ERROR("hipCtxPopCurrent failed in prepareCustomKernel.");
    return InferStatus(holoinfer_code::H_ERROR, "Data processor, hipCtxPopCurrent failed.");
  }
  return InferStatus();
}

}  // namespace inference
}  // namespace holoscan
