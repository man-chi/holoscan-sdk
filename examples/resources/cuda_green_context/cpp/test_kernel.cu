/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

// matrix multiplication
__global__ void matrix_multiply(float* A, float* B, float* C, int N) {
  // Calculate the row and column indices for the current thread
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Initialize the sum for the current element of C
  float sum = 0.0f;

  // Perform the matrix multiplication for the current element
  if (row < N && col < N) {
    for (int i = 0; i < N; ++i) {
      sum += A[row * N + i] * B[i * N + col];
    }
    C[row * N + col] = sum;
  }
}

void asyncLaunchMatrixMultiplyKernel(float* A, float* B, float* C, int N, hipStream_t stream) {
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
  matrix_multiply<<<numBlocks, threadsPerBlock, 0, stream>>>(A, B, C, N);
}
